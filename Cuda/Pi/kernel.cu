#include "hip/hip_runtime.h"
﻿
#include "hip/hip_runtime.h"
#include ""

#include <iostream>
#include <limits>

using ullong = unsigned long long;

constexpr ullong N_BLOCKS =     10u;
constexpr ullong N_THREADS =    1'000u;
constexpr ullong ITERATIONS =   10'000'000ul;

#define CUDA_CALL(x) \
    do { if((x)!=hipSuccess) { \
    printf("Error at %s:%d\n",__FILE__,__LINE__);\
    return EXIT_FAILURE;}} while(0)


__device__ unsigned RandomUInt(unsigned mod) {
    static unsigned seed = 676767676767676;

    seed = seed * 1103515245 + 123456789 * mod;

    return seed;
}

__global__ void MonteCarlo(unsigned long long* counter) {

    for (unsigned long long i = 0; i < ITERATIONS; i++) {

        double x = (double)RandomUInt(blockIdx.x * blockDim.x + threadIdx.x) / UINT_MAX;
        double y = (double)RandomUInt(blockIdx.x * blockDim.x + threadIdx.x) / UINT_MAX;

        if (x * x + y * y <= 1) {
            atomicAdd(counter, 1);
        }

    }
}

int main() {

    //host memory
    unsigned long long output;

    //device memory
    unsigned long long* deviceCounter = nullptr;

    //setting device memory
    CUDA_CALL(hipMalloc((void**)&deviceCounter, sizeof(unsigned long long)));
    hipMemset(deviceCounter, 0, sizeof(unsigned long long));

    //calling kernel
    MonteCarlo <<<BLOCKS, THREADS>>> (deviceCounter);

    hipDeviceSynchronize();

    //retrieving info from device
    hipMemcpy(&output, deviceCounter, sizeof(unsigned long long), hipMemcpyDeviceToHost);

    std::cout << "total points: " << N_BLOCKS * N_THREADS * ITERATIONS << "\t circle points: " 
        << output << std::endl << std::endl;

    //outputing results
    printf("pi: %Lf", ((long double)output / N_BLOCKS * N_THREADS * ITERATIONS) * 4);

 

    //cleanup
    hipFree(deviceCounter);

    return 0;
}


